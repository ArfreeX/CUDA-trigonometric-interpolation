#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <math.h>


#define BLOCK_SIZE 8 // needs to be checked for proper values

__global__ void computeA(double* arrayA, double* arrayOfPoints, double* argumentsArray, int degree, int size)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x < degree + 1)
	{
		double sum = 0;

		for (int i = 0; i < size; i++)	// petla do zrownoleglenia
		{
			sum += (arrayOfPoints[size + i] * cos(x * argumentsArray[i]));
		}

		arrayA[x] = 2.0 / size * sum;
	}
}


__global__ void computeB(double* arrayB, double* arrayOfPoints, double* argumentsArray, int degree, int size)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x < degree + 1)
	{
		double sum = 0;

		for (int i = 0; i < size; i++)	// petla do zrownoleglenia
		{
			sum += (arrayOfPoints[size + i] * sin(x * argumentsArray[i]));
		}

		arrayB[x] = 2.0 / size * sum;
	}
}

void trigInterpolation(double *arrayOfPoints, int size)
{
	const long double PI = std::acos(-1.L);
	int degree = size / 2;
	
	double *arrayA, *arrayB, *argumentsArray;
	double *d_arrayOfPoints, *d_arrayA, *d_arrayB, *d_argumentsArray;

	arrayA = new double[degree+1];
	arrayB = new double[degree+1];

	argumentsArray = new double[size];

	


	int size_bytes = size * 2 * sizeof(double);		// number of bytes allocated on device mem
	int size_bytes_degree = degree + 1 * sizeof(double);
	int size_bytes_args = size * sizeof(double);
	dim3 threadsPerBlock(BLOCK_SIZE);
	dim3 numBlocks((2 * size + BLOCK_SIZE - 1) / BLOCK_SIZE);


	// Cuda allocation
	auto err = hipMalloc(&d_arrayOfPoints, size_bytes);
	if (err != hipSuccess) { std::cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << std::endl; }

	err = hipMalloc(&d_arrayA, size_bytes_degree);
	if (err != hipSuccess) { std::cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << std::endl; }

	err = hipMalloc(&d_arrayB, size_bytes_degree);
	if (err != hipSuccess) { std::cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << std::endl; }

	err = hipMalloc(&d_argumentsArray, size_bytes/2);
	if (err != hipSuccess) { std::cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << std::endl; }




	// Arguments * 2Pi/size && filling values
	for (int i = 0; i < size; ++i)	
	{
		argumentsArray[i] = ((2*PI)/(double)size) * arrayOfPoints[i];
	}




	// Cuda data copy
	err = hipMemcpy(d_arrayOfPoints, arrayOfPoints, size_bytes, hipMemcpyHostToDevice);
	if (err != hipSuccess) { std::cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << std::endl; }

	err = hipMemcpy(d_argumentsArray, argumentsArray, size_bytes_args, hipMemcpyHostToDevice);
	if (err != hipSuccess) { std::cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << std::endl; }

	err = hipMemcpy(d_arrayA, arrayA, size_bytes_degree, hipMemcpyHostToDevice);
	if (err != hipSuccess) { std::cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << std::endl; }

	err = hipMemcpy(d_arrayB, arrayB, size_bytes_degree, hipMemcpyHostToDevice);
	if (err != hipSuccess) { std::cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << std::endl; }

	// Main algorithm code



	//============== CUDA ==============

	// Parameters A
	computeA << < numBlocks, threadsPerBlock >> > (d_arrayA, d_arrayOfPoints, d_argumentsArray, degree, size);

	err = hipMemcpy(arrayA, d_arrayA, size_bytes_degree, hipMemcpyDeviceToHost);
	if (err != hipSuccess) { std::cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << std::endl; }



	// Parameters B	
	computeB << < numBlocks, threadsPerBlock >> > (d_arrayB, d_arrayOfPoints, d_argumentsArray, degree, size);

	err = hipMemcpy(arrayB, d_arrayB, size_bytes_degree, hipMemcpyDeviceToHost);
	if (err != hipSuccess) { std::cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << std::endl; }


	//==================================
	// Results check

	std::cout << "\n\n";
	for (int i = 0; i < 3; i++)
		std::cout << arrayA[i] << "\n";
	for (int i = 0; i < 3; i++)
		std::cout << arrayB[i] << "\n";
	//==================================


	// wypisywanie algorytmu
	std::cout << "\nUzyskana funkcja interpolujaca \n\nG(x) = ";
	std::cout << arrayA[0] / 2.0;
	
	for (int i = 1; i < degree; i++)
	{
		if (arrayA[i] != 0)
		{
			std::cout << " + " << arrayA[i] << "*" << "cos( ";
			if (i == 1)	
				std::cout << "x )";
			else
				std::cout << i << "x )";
		}

		if (arrayB[i] != 0)
		{
			std::cout << " + " << arrayB[i] << "*" << "sin( ";
			if (i == 1)	
				std::cout << "x )";
			else
				std::cout << i << "x )";
		}
	}

	if (size % 2)
		std::cout << " + " << arrayA[degree]<< "*cos( " << degree << "x ) + " << arrayB[degree] << "sin( " << degree << "x )\n\n";
	else
		std::cout << " + " << arrayA[degree] / 2.0 << "*cos( " << degree << "x )\n\n";
	
	
	delete[] arrayA;
	delete[] arrayB;
	delete[] argumentsArray;
	hipFree(d_argumentsArray);
	hipFree(d_arrayA);
	hipFree(d_arrayB);
	hipFree(d_arrayOfPoints);
}

void showMatrix(double *array, int size)
{
	if (size > 0)
	{
		std::cout << "\n[  x ]: ";
		for (int i = 0; i < size; i++)
			std::cout << array[i] << "\t";

		std::cout << "\n[f(x)]: ";
		for (int i = size; i < size * 2; i++)
			std::cout << array[i] << "\t";
	}
}

double* readData(double* array, int &size)
{
	CFileStream file;
	std::cout << "Insert file path\n";
	file.openFile();
	array = file.readData(array, size);
	return array;
}

int main()
{
	double * arrayOfPoints = nullptr;
	int size = 0;
	arrayOfPoints = readData(arrayOfPoints, size);

	showMatrix(arrayOfPoints, size);

	trigInterpolation(arrayOfPoints, size);
	
	system("PAUSE");
	delete[] arrayOfPoints;
	return 0;
}